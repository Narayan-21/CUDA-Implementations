// Cuda device selection based on some prop
// Task done - Choose the closest device from all available devices, device closest to compute capability of 1.3

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_error_handler.h"

int main(void) {
    hipDeviceProp_t prop;
    int dev;

    HANDLE_ERROR(hipGetDevice(&dev));
    printf("ID of the current CUDA devide: %d\n", dev);

    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 1;
    prop.minor = 3;

    HANDLE_ERROR(hipChooseDevice(&dev, &prop));
    printf("ID of CUDA device closest to revision 1.3: %d\n", dev);

    HANDLE_ERROR(hipSetDevice(dev));
}