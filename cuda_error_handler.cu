#include "cuda_error_handler.h"

void HandleError(hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
        std::cerr << hipGetErrorString(err)
                  << " in " << file
                  << " at line " << line
                  << std::endl;
        exit(EXIT_FAILURE);
    }
}