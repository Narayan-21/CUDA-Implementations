#include "hip/hip_runtime.h"
// Vector Sum
#include <stdio.h>
#include "cuda_error_handler.h"
#include <hip/hip_runtime.h>

#define N 10

__global__ void add(int *a, int *b, int *c) {
    int tid = blockIdx.x;
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    };
}

int main(void) {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    HANDLE_ERROR(hipMalloc((void**)&dev_a, N*sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, N*sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, N*sizeof(int)));

    // filling the arrays a and b on the CPU here, but there is no specific need to do so. This can also be done on GPU.
    for (int i=0; i<N; i++){
        a[i] = -i;
        b[i] = i*i;
    };

    HANDLE_ERROR(hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice));

    add<<<N,1>>>(dev_a, dev_b, dev_c);
    HANDLE_ERROR(hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost));

    for (int i=0; i<N; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}