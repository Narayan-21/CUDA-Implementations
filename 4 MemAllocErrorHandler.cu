// Custom cuda error handler for memory allocation

#include <iostream>
#include <hip/hip_runtime.h>


#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

static void HandleError(hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
        std::cerr << hipGetErrorString(err) << " in " << file << " at line " << line << std::endl;
        exit(EXIT_FAILURE);
    }
}

__global__ void add(int a, int b, int *c) {
    *c = a+b;
}

int main(void) {
    int c;
    int *dev_c;

    // GPU memory allocation
    HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int)));

    // Launch Kernel
    add<<<1,1>>>(2,7,dev_c);

    HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));

    printf("2 + 7 = %d\n", c);

    hipFree(dev_c);
    return 0;
}
