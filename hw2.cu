#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 100

__global__ void fun() {
    printf("%d\n", threadIdx.x);
}

int main() {
    fun<<<1, N>>>();
    hipDeviceSynchronize();
    return 0;
}