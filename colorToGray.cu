#include <stdio.h>
#include <hip/hip_runtime.h>

// Number of channels in RGB image
#define CHANNELS 3

// CUDA kernel: convert RGB image to grayscale
__global__
void colorToGrayscaleConversion(unsigned char *Pout,
                                unsigned char *Pin,
                                int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        // Linear index for grayscale image
        int grayOffset = row * width + col;

        // Linear index for RGB image (3 channels per pixel)
        int rgbOffset = grayOffset * CHANNELS;

        unsigned char r = Pin[rgbOffset];     // Red
        unsigned char g = Pin[rgbOffset + 1]; // Green
        unsigned char b = Pin[rgbOffset + 2]; // Blue

        // Weighted sum to convert to grayscale
        Pout[grayOffset] = (unsigned char)(0.21f * r +
                                           0.71f * g +
                                           0.07f * b);
    }
}

int main() {
    // Example dimensions (1500 x 2000 image)
    int width = 2000;
    int height = 1500;
    int numPixels = width * height;

    // Host memory allocation
    unsigned char *h_in  = (unsigned char*)malloc(numPixels * CHANNELS);
    unsigned char *h_out = (unsigned char*)malloc(numPixels);

    // (Here you would normally load image data into h_in)
    // For demo: just fill with random data
    for (int i = 0; i < numPixels * CHANNELS; i++) {
        h_in[i] = (unsigned char)(rand() % 256);
    }

    // Device memory allocation
    unsigned char *d_in, *d_out;
    hipMalloc((void**)&d_in,  numPixels * CHANNELS);
    hipMalloc((void**)&d_out, numPixels);

    // Copy input image to device
    hipMemcpy(d_in, h_in, numPixels * CHANNELS, hipMemcpyHostToDevice);

    // Configure blocks and grids
    dim3 dimBlock(16, 16, 1);
    dim3 dimGrid((width  + dimBlock.x - 1) / dimBlock.x,
                 (height + dimBlock.y - 1) / dimBlock.y,
                 1);

    // Launch kernel
    colorToGrayscaleConversion<<<dimGrid, dimBlock>>>(d_out, d_in, width, height);

    // Copy result back to host
    hipMemcpy(h_out, d_out, numPixels, hipMemcpyDeviceToHost);

    // (Here you would normally save h_out as a grayscale image)
    printf("Conversion done! Example output pixel[0] = %d\n", h_out[0]);

    // Free memory
    free(h_in);
    free(h_out);
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}
