#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 16

__global__ void MatrixMulDynamic(float* M, float* N, float* P, int Width) {
    extern __shared__ float Mds_Nds[];

    float* Mds = Mds_Nds;
    float* Nds = &Mds_Nds[TILE_WIDTH * TILE_WIDTH]; // Start right after the elements used by Mds

    int tx = threadIdx.x; int bx = blockIdx.x;
    int ty = threadIdx.y; int by = blockIdx.y;

    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    float PValue = 0.0f;
    for (int ph = 0; ph < ceil(Width / (float)TILE_WIDTH); ++ph) {
        if ((Row < Width) && (ph * TILE_WIDTH + tx) < Width)
            Mds[ty * TILE_WIDTH + tx] = M[Row * Width + ph * TILE_WIDTH + tx];
        else
            Mds[ty * TILE_WIDTH + tx] = 0.0f;

        if ((ph * TILE_WIDTH + ty) < Width && Col < Width)
            Nds[ty * TILE_WIDTH + tx] = N[(ph * TILE_WIDTH + ty) * Width + Col];
        else
            Nds[ty * TILE_WIDTH + tx] = 0.0f;

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; ++k) {
            PValue += Mds[ty * TILE_WIDTH + k] * Nds[k * TILE_WIDTH + tx];
        }
        __syncthreads();
    }
    if (Row < Width && Col < Width) {
        P[Row * Width + Col] = PValue;
    }
};

void randomInit(float* data, int size) {
    for (int i = 0; i < size; ++i) {
        data[i] = (float)(rand() % 10);
    }
}

int main() {
    int Width = 64;
    int size = Width * Width;
    size_t memSize = size * sizeof(float);

    float* h_M = (float*)malloc(memSize);
    float* h_N = (float*)malloc(memSize);
    float* h_P = (float*)malloc(memSize);

    randomInit(h_M, size);
    randomInit(h_N, size);

    float *d_M, *d_N, *d_P;
    hipMalloc((void**)&d_M, memSize);
    hipMalloc((void**)&d_N, memSize);
    hipMalloc((void**)&d_P, memSize);

    hipMemcpy(d_M, h_M, memSize, hipMemcpyHostToDevice);
    hipMemcpy(d_N, h_N, memSize, hipMemcpyHostToDevice);

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dimGrid((Width + TILE_WIDTH - 1) / TILE_WIDTH,
                 (Width + TILE_WIDTH - 1) / TILE_WIDTH, 1);

    size_t sharedMemSize = 2 * TILE_WIDTH * TILE_WIDTH * sizeof(float);
    MatrixMulDynamic<<<dimGrid, dimBlock, sharedMemSize>>>(d_M, d_N, d_P, Width);
    hipMemcpy(h_P, d_P, memSize, hipMemcpyDeviceToHost);

    printf("Result matrix P (top-left 8x8 block):\n");
    for (int i = 0; i < 8 && i < Width; i++) {
        for (int j = 0; j < 8 && j < Width; j++) {
            printf("%6.1f ", h_P[i * Width + j]);
        }
        printf("\n");
    }

    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);

    free(h_M);
    free(h_N);
    free(h_P);

    return 0;
}