#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void kernel(int *c) {
    *c = (*c) * (*c);
}

int main(void) {
    // Host & device pointers
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    printf("Number of CUDA devices: %d\n", deviceCount);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Running on GPU: %s\n", prop.name);

    int *h_c, *d_c;
    int value = 5;

    h_c = (int*)malloc(sizeof(int));
    *h_c = value;

    // allocate memory on device
    hipMalloc((void**)&d_c, sizeof(int));

    // copy data from host to device
    hipMemcpy(d_c, h_c, sizeof(int), hipMemcpyHostToDevice);

    dim3 grid_size(1);
    dim3 block_size(1);

    kernel<<<grid_size, block_size>>>(d_c);

    hipMemcpy(h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
    printf("Result = %d\n", *h_c);
    hipFree(d_c);
    free(h_c);
    return 0;
}