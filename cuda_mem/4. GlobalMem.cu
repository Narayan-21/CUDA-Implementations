// Global Memory - global memory, grid scope and application lifetime


#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ int globalVar;

__global__ void kernelGlobal() {
    globalVar = threadIdx.x;
}

int main() {
    kernelGlobal<<<1,8>>>();
    hipDeviceSynchronize();

    int hVal;
    hipMemcpyFromSymbol(&hVal, HIP_SYMBOL(globalVar), sizeof(int), hipMemcpyDeviceToHost);
    printf("Final value of globalVar = %d\n", hVal);
    return 0;
}