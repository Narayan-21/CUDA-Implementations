// __threadfence_system() - visibility to host and device.


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void threadFenceSystem(int *data) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    data[tid] = tid;
    __threadfence_system();
};

int main() {
    int *d_data;
    int h_data[256];

    hipMalloc(&d_data, 256*sizeof(int));
    threadFenceSystem<<<1, 256>>>(d_data);
    hipDeviceSynchronize();

    hipMemcpy(h_data, d_data, 256*sizeof(int), hipMemcpyDeviceToHost);

    for (int i =0; i<256; i++)
        printf("Data[%d] = %d\n", i, h_data[i]);
    
    hipFree(d_data);
    return 0;
}