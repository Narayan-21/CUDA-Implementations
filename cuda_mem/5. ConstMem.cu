// 5. Constant Memory - constant memory, grid scope, application lifetime
// Only the host can modify the constant variable.


#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ __constant__ int constVar;   // constant in device memory (read-only)

__global__ void kernelConstant() {
    printf("Thread %d reads constVar=%d\n", threadIdx.x, constVar);
}

int main() {
    int hVal = 99;
    hipMemcpyToSymbol(HIP_SYMBOL(constVar), &hVal, sizeof(int)); // copy from host to constant memory

    kernelConstant<<<1, 4>>>();
    hipDeviceSynchronize();
    return 0;
}
