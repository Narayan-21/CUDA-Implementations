// 5. Constant Memory - constant memory, grid scope, application lifetime


#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ __constant__ int constVar;   // constant in device memory

__global__ void kernelConstant() {
    printf("Thread %d reads constVar=%d\n", threadIdx.x, constVar);
}

int main() {
    int hVal = 99;
    hipMemcpyToSymbol(HIP_SYMBOL(constVar), &hVal, sizeof(int)); // copy from host to constant memory

    kernelConstant<<<1, 4>>>();
    hipDeviceSynchronize();
    return 0;
}
